
#include "cuda_wrapper.h"

ErrorType CudaMallocManaged(void * pointer_managed, size_t buffer_size)
{   
    hipMallocManaged((void **)&pointer_managed, buffer_size);
    return;
}

ErrorType CudaSetDevice(int device_index){
    hipSetDevice(device_index);
    return ;
}

ErrorType CudaFree(void * pointer_managed){
    hipFree(pointer_managed);
    return;
}