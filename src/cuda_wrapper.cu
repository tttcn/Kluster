
#include "cuda_wrapper.h"

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "debug_tool.h"

namespace Kluster {
ErrorType CudaCheck(hipError_t error_code) {
  ErrorType execute_state = NO_ERROR;
  if (error_code != hipSuccess) {
    execute_state = CUDA_ERROR;
    DEBUG("Cuda error name: %s\n", hipGetErrorName(error_code));
    DEBUG("Cuda error description: %s\n", hipGetErrorString(error_code));
  }
  return execute_state;
}

// ErrorType CudaMemPrefetchAsync(void ** pointer_address, size_t buffer_size)
// {
//     return CudaCheck(hipMemPrefetchAsync(pointer_address, buffer_size));
// }

ErrorType CudaMallocManaged(void **pointer_address, size_t buffer_size) {
  return CudaCheck(hipMallocManaged(pointer_address, buffer_size));
}

ErrorType CudaSetDevice(int device_index) {
  return CudaCheck(hipSetDevice(device_index));
}

ErrorType CudaFree(void *pointer_managed) {
  return CudaCheck(hipFree(pointer_managed));
}

// ErrorType CudaMemPrefetchAsync(void * pointer_managed, size_t buffer_size)
// {
//     return
//     CudaCheck(hipMemPrefetchAsync(pointer_managed,buffer_size,0,NULL));
// }
}